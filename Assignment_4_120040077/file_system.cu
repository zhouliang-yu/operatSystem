﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ __managed__ u32 gtime = 0;
__device__ __managed__ u32 gtime_create = 0;
__device__ __managed__ u32 file_start_location = 0;
__device__ __managed__ u32 FCB_position = 4096;
__device__ __managed__ u32 current_FCB_position = 4096;

__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

}



/*
 * my FCB structure
 * |0|1|2|3|4|5|6|7|8|9|10|11|12|13|14|15|16|17|18|19|   20|21   | 22|23 | 24 | 25|26  | 27 |28  |29 | 30|  31|
 * |                    file name                    |       location    |      size        |create_t|modify_t|
 */
struct My_FCB
{
  char file_name[20];
  u32 location;
  u32 size;
  int create_time;
  int modified_time;
}Current_FCB;


__device__ u32 search_FCB(FileSystem *fs, char *s)
{
  int flag_find;
  for (int i = fs->SUPERBLOCK_SIZE; i < fs->FILE_BASE_ADDRESS - 1; i += fs->FCB_SIZE)
  {
  	  flag_find = 0;
	  if (fs->volume[i + 24] == 0 && fs->volume[i + 25] == 0 && fs->volume[i + 26] == 0 && fs->volume[i + 27] == 0) // nothing has been stored
	  {						  // cannot find
		  break;
	  }
	  else
	  {
		  
		  for(int j = 0; j < 20; j++)
		  {
			  if (fs->volume[i + j] != s[j])
			  {

				  flag_find = 1;
				  break;
			  }
		  }
	  }

	  if (flag_find == 0)
	  {
		  return i;
	  }
	  else
	  {
		  continue;
	  }
  }

  return -1;

}

__device__ u32 file_info_store(FileSystem *fs, char *s){
	gtime++;
	gtime_create++;
	current_FCB_position = FCB_position;
	for (int i = 0; i < 20; i++)
	{ // 0-20 stores the file name
		fs->volume[FCB_position + i] = s[i];
	}

	//store the create time
	fs->volume[FCB_position + 28] = gtime_create >> 8;
	fs->volume[FCB_position + 29] = gtime_create & 0x000000FF;

	//store the modified time
	fs->volume[FCB_position + 30] = gtime >> 8;
	fs->volume[FCB_position + 31] = gtime & 0x000000FF;

	//store the start location of block
	fs->volume[FCB_position + 20] = file_start_location >> 24;
	fs->volume[FCB_position + 21] = file_start_location >> 16;
	fs->volume[FCB_position + 22] = file_start_location >> 8;
	fs->volume[FCB_position + 23] = file_start_location;


	//update the time
	//gtime++;
	//gtime_create++;

	//update FCB position
	FCB_position = FCB_position + 32;
	
}


__device__ u32 fs_open(FileSystem *fs, char *s, int op){
  
  u32 file_exist = search_FCB(fs, s);
	/* Implement open operation here */
  if (op == G_READ) { // in the read mode
    if (file_exist == -1) {
      printf("cannot find file in the read mode");
	  return -1;
    }else{ //we find match s
		current_FCB_position = file_exist;
		u32 start_block = (fs->volume[current_FCB_position + 20] << 24) + (fs->volume[current_FCB_position + 21] << 16) + (fs->volume[current_FCB_position + 22] << 8) + (fs->volume[current_FCB_position + 23]);

		return start_block;
	}
  }

  if(op == G_WRITE) {
	  if (file_exist == -1) // if the file doesn't exist create a file in FCB
	  {
		file_info_store(fs, s);
		
		return file_start_location;
	  }else{
		  gtime++;
		  current_FCB_position = file_exist;
		  u32 start_block = (fs->volume[current_FCB_position + 20] << 24) + (fs->volume[current_FCB_position + 21] << 16) + (fs->volume[current_FCB_position + 22] << 8) + (fs->volume[current_FCB_position + 23]);
		 
		 
		  
		  //get the size
		  u32 size = (fs->volume[current_FCB_position + 24]<<24) + (fs->volume[current_FCB_position + 25] <<16) + (fs->volume[current_FCB_position + 26]<<8) + fs->volume[current_FCB_position + 27];
		  //clear the old file content in storage
		  for (int i = 0; i < size; i++)
		  {
			  fs->volume[start_block * fs->FCB_SIZE + i + fs->FILE_BASE_ADDRESS] = 0;
		  }

		  //clear the old file in the superblock because each bit in superblock represent a block in storage
		  for (int i = 0; i < (size - 1) / 32 + 1; i++)
		  {
			  fs->volume[(start_block + i) / 8] = fs->volume[(start_block + i) / 8] - (1 << ((start_block + i) % 8));
		  }

		  //update the modified time
		  fs->volume[current_FCB_position + 30] = gtime >> 8;
		  fs->volume[current_FCB_position + 31] = gtime & 0x000000FF;

		  //update gtime
		  //gtime++;

          //set the mode to write
		  return start_block; //a pointer at super block
	  }
     }
}



__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
	if(fp == -1){
		printf("error\n");
	}
	
	for (int i = 0; i < size; i++){
		output[i] = fs->volume[fp * fs->STORAGE_BLOCK_SIZE + i + fs->FILE_BASE_ADDRESS];
	}
}



__device__ void manage_segmentation(FileSystem * fs, u32 fp, u32 original_size, u32 size)
{
	u32 block_position = fp * 32 + fs->FILE_BASE_ADDRESS;
	u32 new_size = ((original_size - size - 1) / 32 + 1) * 32;
	while ((fs->volume[block_position + new_size] != 0 || (block_position + new_size) % 32 != 0) && block_position + (original_size - size) < fs->STORAGE_SIZE){
		fs->volume[block_position] = fs-> volume[block_position + new_size];
		fs->volume[block_position + new_size] = 0;
		block_position++;
	}

	/** manage the superblock*/
	for (int i =0 ; i < file_start_location /8 + 1; i++) {
		fs->volume[i] = 0;
	}

	file_start_location = file_start_location - ((original_size - size) -1) / 32 - 1;
	u32 file_start_location_q = file_start_location / 8;
	u32 file_start_location_r = block_position % 8;

	for (int i = 0; i < file_start_location_q && i < fs->SUPERBLOCK_SIZE; i ++) {
		fs->volume[i] = 512 - 1;
	}
	for (int i = 0; i < file_start_location_r; i++)
	{
		fs->volume[file_start_location_q] = fs->volume[file_start_location_q] + (1 << i);
	}

	//change FCB
	u32 FCB_block_position;
	for (int i = 4096; i < 36863; i = i + 32)
	{
		if (fs->volume[i + 24] == 0 && fs->volume[i + 25] == 0 && fs->volume[i + 26] == 0 && fs->volume[i + 27] == 0)
		{
			break;
		}
		FCB_block_position = (fs->volume[i + 20] << 24) + (fs->volume[i + 21] << 16) + (fs->volume[i + 22] << 8) + (fs->volume[i + 23]);
		if (FCB_block_position > fp)
		{
			FCB_block_position = FCB_block_position - ((original_size - size) - 1) / 32 - 1;
			fs->volume[i + 20] = FCB_block_position >> 24;
			fs->volume[i + 21] = FCB_block_position >> 16;
			fs->volume[i + 22] = FCB_block_position >> 8;
			fs->volume[i + 23] = FCB_block_position;
		}
	}
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
	/* Implement write operation here */
	
	// check if the file is in the write mood
	// if (((fp & 0xf0000000) >> 30) != G_WRITE)
	// {
	// 	printf("no writing allowed \n");
	// }

	if(size > fs->MAX_FILE_NUM) {
		printf("incorrect error\n");
		return -1;
	}

	fp &= 0x0fffffff;
	if (fp == -1){
		printf(" error\n");
	}

	int enough_space = (fs->volume[(fp + (size - 1) / 32) / 8] >> (fp + (size - 1) / 32) % 8) % 2;

	/**get the original file size*/
	// check if there is enough space for it
	u32 original_size = (fs->volume[current_FCB_position + 24] << 24) + (fs->volume[current_FCB_position + 25] << 16) + (fs->volume[current_FCB_position + 26] << 8) + fs->volume[current_FCB_position + 27];

	

	// clear all the contents in storage
	//for (int i = 0; i < original_size; i ++) {
	//	fs->volume[fp * fs->FCB_SIZE + i + fs->FILE_BASE_ADDRESS] = 0;
	//}

	//enough space to write
	if(enough_space == 0) {
		//write in the storage
		for (int i = 0; i < size; i++)
		{
			fs->volume[fp * fs->STORAGE_BLOCK_SIZE + i + fs->FILE_BASE_ADDRESS] = input[i];
		}
		//update the superblock
		for (int i = 0; i < size; i++)
		{
			if (i % 32 == 0) {
				fs->volume[(fp + i / 32) / 8] = fs->volume[(fp + i / 32) / 8] + (1 << ((fp + i / 32) % 8));
			}
		}

		if (int(original_size - size) < 0){
			file_start_location = file_start_location + (-(original_size - size) - 1) / 32 + 1;
		}
		//update the size in FCB
		fs->volume[current_FCB_position + 24] = size >> 24;
		fs->volume[current_FCB_position + 25] = size >> 16;
		fs->volume[current_FCB_position + 26] = size >> 8;
		fs->volume[current_FCB_position + 27] = size;

			if (int(original_size - size) > 0 && original_size != 0 && fp != file_start_location - 1)
		{
			manage_segmentation(fs, fp, original_size, size);
		}

	}
	else{ //DONT have enough space
		if (file_start_location * 32 - 1 + size < fs->SUPERBLOCK_SIZE){
			for (int i = 0; i < size; i ++) {
				fs->volume[file_start_location * 32 + i + fs->FILE_BASE_ADDRESS] = input[i];
			//update the superblock
			if(i % 32 == 0){
				fs->volume[(file_start_location + i / 32) / 8] = fs->volume[(file_start_location + i / 32) / 8] + (1 << ((file_start_location + i / 32) % 8));
			}
			
			//update the FCB
			fs->volume[current_FCB_position + 24] = size >> 24;
			fs->volume[current_FCB_position + 25] = size >> 16;
			fs->volume[current_FCB_position + 26] = size >> 8;
			fs->volume[current_FCB_position + 27] = size;

			//update block position
			fs->volume[i + 20] = file_start_location >> 24;
			fs->volume[i + 21] = file_start_location >> 16;
			fs->volume[i + 22] = file_start_location >> 8;
			fs->volume[i + 23] = file_start_location;
			}
		manage_segmentation(fs, fp, original_size, size);
		}
	}
	


}



__device__ void sort(FileSystem *fs, u32 begin, u32 end, int op) {
	
	if (op == 1) { //by size
		for (int i = begin; i < end; i = i + 32){
			for (int j = begin; j < end + begin - i ; j = j+ 32){
				u32 j_size_pre = (fs->volume[j+24] << 24) + (fs->volume[j + 25] << 16)  + (fs->volume[j + 26] << 8) + (fs->volume[j + 27]);
				u32 j_size_after = (fs->volume[j+24 + 32] << 24) + (fs->volume[j + 25 +32] << 16)  + (fs->volume[j + 26+32] << 8) + (fs->volume[j + 27+32]);
				u32 j_time_pre = (fs->volume[j + 28] << 8) + (fs->volume[j + 29]);
				u32 j_time_after = (fs->volume[j + 28 + 32] << 8) + (fs->volume[j + 29 + 32]);
				if (j_size_pre < j_size_after){
					// swap
					for (int k = 0; i < 32; i++)
					{
						uchar tempt = fs->volume[j + k];
						fs->volume[j + k] = fs->volume[j + k + 32];
						fs->volume[j + k + 32] = tempt;
					}
				}
				if (j_size_after == j_size_pre && j_time_pre > j_time_after){
					// swap
					for (int k = 0; k < 32; k++)
					{
						uchar tempt = fs->volume[j + k];
						fs->volume[j + k] = fs->volume[j + k + 32];
						fs->volume[j + k + 32] = tempt;
					}
				}
			}
		}
	}else{ // by time
		for (int i = begin; i < end; i = i + 32)
		{
			for (int j = begin; j < end + begin - i; j = j + 32)
			{
				u32 j_time_prev = (fs->volume[j + 28] << 8) + (fs->volume[j + 29]);
				u32 j_time_after = (fs->volume[j + 28 + 32] << 8) + (fs->volume[j + 29 + 32]);
				//printf("prev time is: %d\n", j_time_prev );
				//printf("after time is: %d\n", j_time_after );
				//printf("examinater \n");
				if (j_time_prev < j_time_after){
					// swap
					//printf("do we swap \n");
					//printf("yes \n");
					for (int k = 0; k < 32; k++)
					{
						
						uchar tempt = fs->volume[j + k];
						fs->volume[j + k] = fs->volume[j + k + 32];
						fs->volume[j + k + 32] = tempt;
					}
				}
			}
		}
	}

}

__device__ void display(FileSystem*fs, u32 end_point, int op){
	char file_name[20];
	if (op != 0) { // sort by file size
		for (u32 i = 4096; i <= end_point; i = i + 32)
		{
			for (int j = 0; j < 20; j++)
			{
				file_name[j] = fs->volume[i + j];
			}
			u32 size = (fs->volume[i + 24] << 24) + (fs->volume[i + 25] << 16) + (fs->volume[i + 26] << 8) + fs->volume[i + 27];
			printf("%s %d\n", file_name, size);
		}
	}
	else{ //sort by time
		for (u32 i = 4096; i <= end_point; i = i + 32)
		{
			for (int j = 0; j < 20; j++)
			{
				file_name[j] = fs->volume[i + j];
			}
			printf("%s\n", file_name);
		}
	}



}




__device__ void fs_gsys(FileSystem *fs, int op)
{
	/* Implement LS_D and LS_S operation here */
	/** sort by date*/
	u32 end_point;
	for (u32 i = 4096; i < 36863 + 32; i += 32){
		u32 size = (fs->volume[i + 24] << 24) + (fs->volume[i + 25] << 16) + (fs->volume[i + 26] << 8) + fs->volume[i + 27];
		if (size == 0) {
			size = (fs->volume[4096 + 24] << 24) + (fs->volume[4096 + 25] << 16) + (fs->volume[4096 + 26] << 8) + (fs->volume[4096 + 27]);
			end_point = i - 32;
			break;
		}
	 end_point = i - 32;
	}
	
	if (end_point < 4096)
	{
		printf("error: no file in FCB \n");
		return;
	}
	
	if(op != 0) { //sort by size
		printf("---sort by file size---\n");
		sort(fs, 4096, end_point, 1);
		// display(fs, end_point, 1);
	}else{
		printf("---sort by time---\n");
		sort(fs, 4096, end_point, 0);
		// display(fs, end_point, 0);
	}

	char file_name[20];
	if (op != 0)
	{ // sort by file size
		for (u32 i = 4096; i <= end_point; i = i + 32)
		{
			for (int j = 0; j < 20; j++)
			{
				file_name[j] = fs->volume[i + j];
			}
			u32 size = (fs->volume[i + 24] << 24) + (fs->volume[i + 25] << 16) + (fs->volume[i + 26] << 8) + fs->volume[i + 27];
			printf("%s %d\n", file_name, size);
		}
	}
	else
	{ //sort by time
		for (u32 i = 4096; i <= end_point; i = i + 32)
		{
			for (int j = 0; j < 20; j++)
			{
				file_name[j] = fs->volume[i + j];
			}
			printf("%s\n", file_name);
		}
	}
}

__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
	/* Implement rm operation here */
	u32 file_exist = search_FCB(fs, s);
	if (file_exist == -1)
		printf("error : the file is not exist\n");
	else
	{
		current_FCB_position = file_exist;
		//find where the file start from FCB
		u32 FCB_start_block = (fs->volume[current_FCB_position + 20] << 24) + (fs->volume[current_FCB_position + 21] << 16) + (fs->volume[current_FCB_position + 22] << 8) + (fs->volume[current_FCB_position + 23]);

		//find the size of file
		u32 size = (fs->volume[current_FCB_position + 24] << 24) + (fs->volume[current_FCB_position + 25] << 16) + (fs->volume[current_FCB_position + 26] << 8) + fs->volume[current_FCB_position + 27];

		//clear content in storage
		for (int i = 0; i < size; i++)
		{
			fs->volume[FCB_start_block * 32 + i + fs->FILE_BASE_ADDRESS] = 0;
		}

		//clean corresponding superblock
		for (int i = 0; i < (size - 1) / 32 + 1; i++)
		{
			fs->volume[FCB_start_block + i] = 0;
		}

		//clean the FCB
		for (int i = 0; i < 32; i++)
		{
			fs->volume[current_FCB_position + i] = 0;
		}

		manage_segmentation(fs, FCB_start_block,  size, 0);
		
		for (int i = current_FCB_position;i < 36863; i = i + 32){
			if (fs->volume[i + 32 + 24] == 0 && fs->volume[i + 32+25] == 0 && fs->volume[i +32+ 26] == 0 && fs->volume[i +32+ 27] == 0){
				for (int j = 0; j < 32; j ++){
					fs->volume[i + j + 32] = 0;
				}
			}
		}


		FCB_position = FCB_position - 32;
	}
}
